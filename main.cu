#include "hip/hip_runtime.h"
#include <glad/glad.h> 
#include <GLFW/glfw3.h>

#include <iostream>
#include <chrono>
#include <string>
#include <iomanip>

#include "shader.h"
#include "ShaderProgram.h"
#include "fpsCounter.h"
#include "Camera.h"

#define GLM_FORCE_CUDA
#define CUDA_VERSION 101000
#include <glm/glm.hpp>
//#include <glm/vec2.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "cuda_gl_interop.h"

#include "kernel.cuh"


// Initial resolution of window
unsigned int winWidth = 512*2;
unsigned int winHeight = 512*2;

namespace settings {
    bool g_useSuperSampling = true;
    Camera g_camera(glm::dvec3(-5., 0., 0.), 0., 0. );
    int g_iterCap = 2;
    bool g_cursorLocked = false;
}

namespace input {
    glm::dvec2 g_cursorPos; // Center of screen is (0, 0), borders are +- 1;
}

namespace screenState {
    // Resolution of render
    unsigned int hRes = 1920 * 2;
    unsigned int vRes = 1027 * 2; // Supports non-powers of two, but powers of two will be somewhat faster
    unsigned int renderTexture;
    unsigned int displayTexture;
    hipGraphicsResource_t screenCudaResource;
}


void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    winWidth = width;
    winHeight = height;

    hipGraphicsUnregisterResource(screenState::screenCudaResource);

    int superSamplingFactor = settings::g_useSuperSampling ? 2 : 1;

    screenState::hRes = width * superSamplingFactor;
    screenState::vRes = height * superSamplingFactor;
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, screenState::renderTexture);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, screenState::hRes, screenState::vRes, 0, GL_RGBA, GL_FLOAT, NULL);

    glActiveTexture(GL_TEXTURE1);
    glBindTexture(GL_TEXTURE_2D, screenState::displayTexture);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, screenState::hRes, screenState::vRes, 0, GL_RGBA, GL_FLOAT, NULL);

    glViewport(0, 0, width, height);

    hipGraphicsGLRegisterImage(&screenState::screenCudaResource, screenState::renderTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
}

void printVec(glm::dvec3 vec) {
    std::cout << '(' << vec.x << ", " << vec.y << ", " << vec.z << ')';
}

// XMousepos and yMousepos given in normalized coords, [-1, 1)
// Positive X axis is to the right, positive Y axis is upwards on the screen
void processInput(GLFWwindow* window)
{
    glm::dvec2 newMousePos;
    glfwGetCursorPos(window, &(newMousePos.x), &(newMousePos.y));
    
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
        settings::g_camera.walk(glm::dvec3(0., 0., -0.15));
    }
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
        settings::g_camera.walk(glm::dvec3(0., 0., 0.15));
    }
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
        settings::g_camera.walk(glm::dvec3(0.15, 0., 0.));
    }
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
        settings::g_camera.walk(glm::dvec3(-0.15, 0., 0.));
    }
    if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS) {
        settings::g_camera.walk(glm::dvec3(0., 0.15, 0.));
    }
    if (glfwGetKey(window, GLFW_KEY_LEFT_SHIFT) == GLFW_PRESS) {
        settings::g_camera.walk(glm::dvec3(0.0, -0.15, 0.));
    }
    newMousePos.x *= 2. / winWidth;
    newMousePos.x -= 1.;
    newMousePos.y *= -2. / winHeight;
    newMousePos.y += 1.;
    
    glm::dvec2 deltaMouse = newMousePos - input::g_cursorPos;
    if (!settings::g_cursorLocked) {
        settings::g_camera.rotate(-1 * deltaMouse.x, deltaMouse.y);
    }
    input::g_cursorPos = newMousePos;
}

void keyPressCallback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (action == GLFW_PRESS) {
        switch (key) {
        case GLFW_KEY_ESCAPE: glfwSetWindowShouldClose(window, true); break;
        //case GLFW_KEY_S:      settings::g_useSuperSampling = !settings::g_useSuperSampling;
        //                      framebuffer_size_callback(window, winWidth, winHeight); break; // Force resizing of window for supersampling settings;
        case GLFW_KEY_LEFT: settings::g_iterCap--; break;
        case GLFW_KEY_RIGHT: settings::g_iterCap++; break;
        case GLFW_KEY_P: settings::g_cursorLocked = !settings::g_cursorLocked; break;
        }
    }

}

void mouseButtonCallback(GLFWwindow* window, int button, int action, int mods)
{
    return;
}

void scrollCallback(GLFWwindow* window, double xoffset, double yoffset)
{
    ;//settings::g_screenBounds.zoom(settings::g_screenBounds.screenPointToWorld(input::g_cursorPos), pow(0.8, -1. * yoffset));
}

int main(void) {

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);



    GLFWwindow* window = glfwCreateWindow(winWidth, winHeight, "RenderTest", NULL, NULL);

    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }

    glViewport(0, 0, winWidth, winHeight);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetMouseButtonCallback(window, mouseButtonCallback);
    glfwSetKeyCallback(window, keyPressCallback);
    glfwSetScrollCallback(window, scrollCallback);

    //
    float vertices[] = {
         // positions     // Tex coords
         1.00f,  1.00f, 0.0f,  1.f,  1.f, // top right
         1.00f, -1.f, 0.0f,  1.f,  0.f,  // bottom right
        -1.f, -1.f, 0.0f,  0.f,  0.f,  // bottom left
        -1.f,  1.00f, 0.0f,  0.f,  1.f,   // top left 
    };
    unsigned int indices[] = {  
        0, 1, 3,   // first triangle
        1, 2, 3    // second triangle
    };
    
    unsigned int VAO;
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    
    unsigned int VBO;
    glGenBuffers(1, &VBO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(0));
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);
    

    unsigned int EBO;
    glGenBuffers(1, &EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
    

    glGenTextures(1, &screenState::renderTexture);

   
    glBindTexture(GL_TEXTURE_2D, screenState::renderTexture);
    {
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
        //float borderColor[] = { 1.0f, 1.0f, 0.0f, 1.0f };
        //glTexParameterfv(GL_TEXTURE_2D, GL_TEXTURE_BORDER_COLOR, borderColor);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, screenState::renderTexture);


        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, screenState::hRes, screenState::vRes, 0, GL_RGBA, GL_FLOAT, NULL);

        glBindImageTexture(0, screenState::renderTexture, 0, GL_FALSE, 0, GL_WRITE_ONLY, GL_RGBA32F);
    }
    glGenTextures(1, &screenState::displayTexture);
    glBindTexture(GL_TEXTURE_2D, screenState::displayTexture);
    {
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);

        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

        glActiveTexture(GL_TEXTURE1);
        glBindTexture(GL_TEXTURE_2D, screenState::displayTexture);


        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, screenState::hRes, screenState::vRes, 0, GL_RGBA, GL_FLOAT, NULL);

        glBindImageTexture(1, screenState::displayTexture, 0, GL_FALSE, 0, GL_WRITE_ONLY, GL_RGBA32F);
    }

    // Vertex and fragment shaders are pretty much empty, and just pass through vertex/texture coord data
    Shader vertex("doNothing.vert", GL_VERTEX_SHADER);
    Shader fragment("doNothing.frag", GL_FRAGMENT_SHADER);
    ShaderProgram renderProg;
    renderProg.attach(vertex);
    renderProg.attach(fragment);
    renderProg.link();

    Shader frameBlendShader("frameBlend.comp", GL_COMPUTE_SHADER);
    ShaderProgram frameBlendProg;
    frameBlendProg.attach(frameBlendShader);
    frameBlendProg.link();

    
    

    input::g_cursorPos;

    FpsCounter fpsCounter;
    framebuffer_size_callback(window, winWidth, winHeight); // Force resizing of window for supersampling settings;

    hiprandState* cuRandPtr = setupRaytracer(screenState::hRes, screenState::vRes);

    while (!glfwWindowShouldClose(window))
    {
        fpsCounter.update(glfwGetTime());
        std::stringstream title;
        title << "Raytracer - FPS: " <<  std::setprecision(0) << std::setiosflags(std::ios::fixed) << fpsCounter.getFPS();
        glfwSetWindowTitle(window, title.str().c_str());

        // Input (updates global variable for mouse position)
        processInput(window);

        glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        hipGraphicsMapResources(1, &screenState::screenCudaResource);
        {
            hipArray_t viewCudaArray;
            hipGraphicsSubResourceGetMappedArray(&viewCudaArray, screenState::screenCudaResource, 0, 0);
            hipResourceDesc viewCudaArrayResourceDesc;
            {
                viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
                viewCudaArrayResourceDesc.res.array.array = viewCudaArray;
            }
            hipSurfaceObject_t viewCudaSurfaceObject;
            hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc);
            {
                traceImage<<<screenState::hRes*screenState::vRes / 128 + 1, 128>>>(viewCudaSurfaceObject, screenState::hRes, screenState::vRes, settings::g_iterCap, settings::g_camera.getPos(), settings::g_camera.getLookDir(), settings::g_camera.getLookU(), settings::g_camera.getLookR(), cuRandPtr);
            }
            hipDestroySurfaceObject(viewCudaSurfaceObject);
        }
        hipGraphicsUnmapResources(1, &screenState::screenCudaResource);

        hipStreamSynchronize(0);


        frameBlendProg.use();
        frameBlendProg.setUvec2("resolution", screenState::hRes, screenState::vRes);

        glDispatchCompute((screenState::hRes + 15) / 16, (screenState::vRes + 15) / 16, 1); // For local work group size 16. Ensures entire texture is written to

        glMemoryBarrier(GL_TEXTURE_FETCH_BARRIER_BIT);

        // Render screen-sized quad

        renderProg.use();
        //renderProg.setInt("textureVar", 1);
        glBindTexture(GL_TEXTURE_2D, screenState::displayTexture);
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
        glBindTexture(GL_TEXTURE_2D, 0);

        // End drawing current frame
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glfwTerminate();
    return 0;
}