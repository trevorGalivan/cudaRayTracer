#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define GLM_FORCE_CUDA
#define CUDA_VERSION 101000
#include <glm/glm.hpp>

#include <ctime>

__device__
const float pi = 3.1415962;



using vec3 = glm::vec3;
using vec4 = glm::vec4;
typedef struct Ray{
    vec3 pos;
    vec3 dir;
    __device__
    Ray(vec3 pos_, vec3 dir_) : pos(pos_), dir(dir_) {}
}Ray;

typedef struct Material { // REPLACE with model that accounts for glossiness, and diffuse reflection
    float roughness;
    float metalness;
    float emmisiveness;
    float IOR;
    vec3 colour;
    __device__
    Material() {};
    __device__
    Material(float roughness_, float metalness_, float emmisiveness_, float IOR_, vec3 colour_) : roughness(roughness_), metalness(metalness_), emmisiveness(emmisiveness_), IOR(IOR_), colour(colour_) {}
}Material;

typedef struct Intersection {
    vec3 pos;
    vec3 normal;
    Material mat;
    __device__
    Intersection(vec3 pos_, vec3 normal_, Material mat_) : pos(pos_), normal(normal_), mat(mat_) {}
}Intersection;

typedef struct Sphere {
    vec3 pos;
    Material mat;
    float radius;
    __device__
    Sphere() {};
    __device__
    Sphere(vec3 pos_, Material mat_, float radius_) : pos(pos_), mat(mat_), radius(radius_) {}
}Sphere;




__device__
float intersect(Ray ray, vec3 ballPos, float ballRad);

__device__
vec3 trace(const Ray ray, const Sphere scene[], const int numSpheres, const int iterCap, hiprandState* cuRandPtr, int index);

__device__
vec3 getDiffuseRay(hiprandState* globalRandState, vec3 normal, int ind);

__device__
vec3 tonemap(vec3 input, float exposure);

__global__
void setupRNG(hiprandState* state, unsigned long int seed, unsigned int xRes, unsigned int yRes)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < xRes * yRes) {
        hiprand_init(seed, index, 0, &state[index]);
    }
}

__host__
hiprandState* setupRaytracer(unsigned int xRes, unsigned int yRes)
{
    hiprandState* cuRandPtr;
    hipMalloc(&cuRandPtr, xRes * yRes * sizeof(hiprandState));

    srand(time(NULL));

    setupRNG <<<(xRes * yRes) / 128 + 1, 128 >>> (cuRandPtr, rand(), xRes, yRes);

    return cuRandPtr;
}

__global__
void traceImage(hipSurfaceObject_t output, unsigned int xRes, unsigned int yRes, int iterCap, vec3 camPos, vec3 camDir, vec3 camUp, vec3 camRight, hiprandState* cuRandPtr)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //int stride = blockDim.x * gridDim.x;

    if(index < xRes * yRes)
    {
        int x = index % xRes;
        int y = index / xRes;

        
        glm::vec2 normalized = glm::vec2(x, y) / glm::vec2(xRes, yRes) * 2.f - 1.f; // [0, 1)

        vec3 rayDirection = camDir * 2.f + camUp * normalized.y + camRight * normalized.x;
        rayDirection = normalize(rayDirection);

        Ray baseRay = { camPos, rayDirection };


        const Sphere scene[9] = { Sphere(vec3(0.f), Material(1.f, 0.f, 0.f, 2.5f,  vec3(0.2f, 0.9f, 0.2f)), 1.f),
                                  Sphere(vec3(3.f), Material(0.5f, 1.f, 0.0f, 1.5f, vec3(0.9f, 0.9f, 0.2f)), 0.5f),
                                  Sphere(vec3(1.5f), Material(1.f, 0.f, 0.f, 1.1f, vec3(0.2f, 0.9f, 0.9f)), 1.f),
                                  Sphere(vec3(6.f), Material(1.f, 0.f, 0.f, 1.1f, vec3(0.9f, 0.9f, 0.1f)), 1.f),
                                  Sphere(vec3(1.f, -5.f, 1.f), Material(0.1f, 0.f, 1.5f, 1.5f, vec3(0.9f, 0.2f, 0.2f)), 3.9f),
                                  Sphere(vec3(1.f, 5.f, 10.f), Material(0.1f, 0.f, 2.5f, 1.5f, vec3(0.3f, 0.5f, 0.8f)), 1.5f),
                                  Sphere(vec3(-1.f, 0.f, 5.f), Material(1.f, 1.f, 0.f, 2.5f, vec3(0.2f, 0.2f, 0.9f)), 1.f),
                                  Sphere(vec3(0.f, -50010.f, 0.f), Material(0.1f, 0.0f, 0.f, 1.1f, vec3(0.25f, 0.2f, 0.25f)), 50000.f),
                                  Sphere(vec3(0.f, 0.f, -50010.f), Material(0.1f, 1.0f, 0.f, 7.1f, vec3(0.5f, 0.5f, 0.5f)), 50000.f)};

        float4 data = *(reinterpret_cast<float4*> (&vec4( tonemap( trace(baseRay, scene, 9, iterCap, cuRandPtr, index), 1.f ), 1) ));

        surf2Dwrite(data, output, x * sizeof(float4), y);
    }


}

__device__
vec3 tonemap(vec3 input, float exposure)
{
    const float a = 2.51f;
    const float b = 0.03f;
    const float c = 2.43f;
    const float d = 0.59f;
    const float e = 0.14f;
    return exposure * (input * (a * input + b)) / (input * (c * input + d) + e);
}

__device__
float intersect(const Ray ray, const vec3 ballPos, const float ballRad)
{
    vec3 oc = ray.pos - ballPos;
    //float a = dot(rayDirection, rayDirection);
    float b = 2.0 * dot(oc, ray.dir);
    float c = dot(oc, oc) - ballRad * ballRad;
    float discriminant = b * b - 4.f * c;
    return (discriminant > 0.f && -1.f * sqrt(discriminant) - b > 0.f) ? (-1.f * sqrt(discriminant) - b) / (2.f) : 1000000000.f;//dot((camPosition + ( ((-1*sqrt(discriminant) - b) / (2.*a)) * rayDirection) - ballPos), vec3(0., 1., 0) ): 0.1;
}

__device__
vec3 trace(const Ray ray, const Sphere scene[], const int numSpheres, const int iterCap, hiprandState* cuRandPtr, int index)
{
    if (iterCap == 0) {
        return vec3(0.f);
    }

    const Material skyMaterial = { 0.f, 0.f, 0.f, 1.f, vec3{0.0f, 0.0f, 0.0f} };

    Sphere nearestHit;
    float hitDist = 10000000.f;
    float recentHit;

    for (int i = 0; i < numSpheres; i++) {
        if ((recentHit = intersect(ray, scene[i].pos, scene[i].radius)) < hitDist) {
            hitDist = recentHit;
            nearestHit = scene[i];
        }
    }

    if (hitDist == 10000000.f) {
        return skyMaterial.colour;
    }

    vec3 position = ray.dir * hitDist + ray.pos;

    vec3 normal = glm::normalize(position - nearestHit.pos);

    float R = pow((1.f - nearestHit.mat.IOR) / (1.f + nearestHit.mat.IOR), 2.f);
    float fresnel = R + (1.f - R) * pow( (1.f + glm::dot(ray.dir, normal) ) , 5.f);
    
    vec3 finalColour = fresnel * (vec3(1.f) * (1.f - nearestHit.mat.metalness) + nearestHit.mat.colour * nearestHit.mat.metalness) * trace(Ray(position, glm::reflect(ray.dir, normal)), scene, numSpheres, iterCap - 1, cuRandPtr, index);
    finalColour += (1.f - fresnel) * (1.f - nearestHit.mat.metalness) * nearestHit.mat.colour * trace(Ray(position, getDiffuseRay(cuRandPtr, normal, index)), scene, numSpheres, iterCap-1, cuRandPtr, index); // This line is buggy, but the bug looks really cool
    finalColour += nearestHit.mat.emmisiveness * nearestHit.mat.colour;

    return vec3(finalColour);
}

__device__
vec3 getDiffuseRay(hiprandState* globalRandState, vec3 normal, int ind)
{
    // Generates two random angles
    hiprandState localState = globalRandState[ind];
    float horiz = hiprand_uniform(&localState) * 2.f * pi; // (0, 2*pi]
    float vert  = hiprand_uniform(&localState)  * pi; // (0, pi]
    globalRandState[ind] = localState;
    // Creates spherical distribution of unit vectors
    vec3 diffuse(sin(vert) * sin(horiz), cos(vert), sin(vert) * cos(horiz));

    // Converts to hemispherical distribution in direction of normal vector
    return glm::dot(diffuse, normal) > 0.f ? diffuse : -1.f * diffuse;
}